#include "hip/hip_runtime.h"
#include "mgn_node_pipe.cuh"
#include "mgn_node_mlp0s0.cuh"
#include "mgn_node_mlp0s1.cuh"
#include "mgn_node_mlp0s2.cuh"
#include "mgn_node_mlp1s0.cuh"
#include "mgn_node_mlp2s0.cuh"

using Block = cutlass::gemm::GemmShape<MgnNodeMlp::mblk, MgnNodeMlp::d, 128>;
using Mlp0Stage0 = Mlp0s0<Block, MgnNodeMlp::num_warps>;
using Mlp0Stage1 = Mlp0s1<Block, MgnNodeMlp::num_warps>;
using Mlp0Stage2 = Mlp0s2<Block, MgnNodeMlp::num_warps>;
using Mlp1Stage0 = Mlp1s0<Block, MgnNodeMlp::num_warps>;
using Mlp2Stage0 = Mlp2s0<Block, MgnNodeMlp::num_warps>;


const size_t max_smem = std::max({
    Mlp0Stage0::smem_bytes,
    Mlp0Stage1::smem_bytes,
    Mlp0Stage2::smem_bytes,
    Mlp1Stage0::smem_bytes,
    Mlp2Stage0::smem_bytes
});

__device__ void mlp0_sm0(half *smem, MgnNodeMlp *prob, size_t row) {
    Mlp0Stage0 pipe(smem, prob->mi, threadIdx.x, threadIdx.y);
    pipe.run(&prob->w1[0][0][0], &prob->in[0][row * prob->mi][0], prob->q1[row][0]);
}

__device__ void mlp0_sm1(half *smem, MgnNodeMlp *prob, size_t row) {
    Mlp0Stage1 pipe(smem, prob->mi, threadIdx.x, threadIdx.y);
    pipe.run(&prob->w1[1][0][0], &prob->in[1][row * prob->mi][0], prob->q1[row][0], prob->q1[row][1]);
}

__device__ void mlp0_sm2(half *smem, MgnNodeMlp *prob, size_t row) {
    Mlp0Stage2 pipe(smem, prob->mi, threadIdx.x, threadIdx.y);
    pipe.run(&prob->w1[2][0][0], &prob->in[2][row * prob->mi][0], prob->q1[row][1], prob->q12[row]);
}

__device__ void mlp1_sm0(half *smem, MgnNodeMlp *prob, size_t row) {
    Mlp1Stage0 pipe(smem, prob->mi, threadIdx.x, threadIdx.y);
    pipe.run(&prob->w2[0][0], prob->q12[row], prob->q23[row]);
}

__device__ void mlp2_sm0(half *smem, MgnNodeMlp *prob, size_t row) {
    Mlp2Stage0 pipe(smem, prob->mi, threadIdx.x, threadIdx.y);
    pipe.run(&prob->w3[0][0], prob->q23[row], &prob->out[row * prob->mi][0]);
}

template<typename QT>
__device__ void consume_dummy(QT& q, size_t num_iters) {

    for (size_t i = 0; i < num_iters; i++) {
        q.read_wait(i);
        q.read_commit(i);
    }
}


__global__ void kernel(MgnNodeMlp * prob) {
    extern __shared__ half smem[];

    size_t pipe_col = blockIdx.x;
    size_t pipe_row = blockIdx.y;

    switch (pipe_col) {
        case 0:
            mlp0_sm0(smem, prob, pipe_row);
            break;
        case 1:
            mlp0_sm1(smem, prob, pipe_row);
            break;
        case 2:
            mlp0_sm2(smem, prob, pipe_row);
            break;
        case 3:
            mlp1_sm0(smem, prob, pipe_row);
            break;
        case 4:
            mlp2_sm0(smem, prob, pipe_row);
            break;

        default: return;
    }
}

int main() {

    cudaErrCheck(hipDeviceSetCacheConfig(hipFuncCachePreferShared));
    cudaErrCheck(hipFuncSetAttribute(reinterpret_cast<const void*>(
        kernel),
        hipFuncAttributePreferredSharedMemoryCarveout,
        cudaSharedmemCarveoutMaxShared));

    cudaErrCheck(hipFuncSetAttribute(reinterpret_cast<const void*>(
        kernel), hipFuncAttributeMaxDynamicSharedMemorySize, max_smem));

    MgnNodeMlp * prob;
    cudaErrCheck(hipMalloc(&prob, sizeof(MgnNodeMlp) + 128));
    // Align prob
    prob = (MgnNodeMlp*)(((size_t)prob + 0x3F) & ~0x3F);

    printf("Init...\n");
    init_prob<<<1, 128>>>(prob);
    cudaErrCheck(hipDeviceSynchronize());

    dim3 grid(5, MgnNodeMlp::mo);
    dim3 block(32, MgnNodeMlp::num_warps);

    printf("SMEM: %lu\n", max_smem);

    printf("Running...\n");
    float time_ms = cuda_time_kernel_ms(
        [&]() {
            kernel<<<grid, block, max_smem>>>(prob);
        }
    );

    printf("gemm took %fms\n", time_ms);

    float flops_v1 =
        2.0f * MgnNodeMlp::m * (3 * MgnNodeMlp::d) * MgnNodeMlp::d +
        2.0f * MgnNodeMlp::m * MgnNodeMlp::d * MgnNodeMlp::d +
        2.0f * MgnNodeMlp::m * MgnNodeMlp::d * MgnNodeMlp::d;
    float gflops_v1 = flops_v1 / (time_ms * 1e6);
    printf("+ GFLOPS: %f\n", gflops_v1);

    return 0;
}
