#include "hip/hip_runtime.h"


#include <torch/extension.h>
#include <ATen/ATen.h>

#include <hip/hip_fp16.h>

#include "pipes.cuh"
#include "pipe_gemm.cuh"
#include "pipe_gemm_bias.cuh"
#include "pipe_gemm_bias_relu.cuh"
#include "pipe_layer_norm.cuh"

#include "utils.cuh"

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

using ProblemShape = cutlass::gemm::GemmShape<128, 128, 128>;

const size_t num_warps = std::max({
    PipeGemm<ProblemShape>::num_warps,
    PipeGemmBias<ProblemShape>::num_warps,
    PipeGemmBiasRelu<ProblemShape>::num_warps
});

template<typename DT, size_t M, size_t D>
struct QueueEntry2D {
    using Element = DT;
    Element buf[M][D];

    __device__ half * as_ptr() { return (half *)buf; }
    __device__ TensorView as_view() { return {as_ptr(), D}; }
    __device__ QueueEntry2D() {}
};

struct MgnFullMlp {
    static const int d = 128;
    static const int n_rows = 40;

    static const int n_mlp_cols = 5;
    static const int n_ln_cols = 5;
    static const int n_cols = n_mlp_cols + n_ln_cols;

    static const int mblk = 128;
    static const int qlen = 2;
    static const int ln_qlen = n_ln_cols + 1;

    int m;

    half * in;
    half * w1;
    half * b1;
    half * w2;
    half * b2;
    half * w3;
    half * b3;
    half * gamma;
    half * beta;
    half * out;

    using QEntry = QueueEntry2D<half, mblk, d>;
    using Queue = MpmcRingQueue<QEntry, qlen, 1, 1>;
    using LayerNormQueue = MpmcRingQueue<QEntry, ln_qlen, 1, 1>;

    struct Queues {
        Queue q01;
        Queue q12;
        Queue q23;
        Queue q34;
        LayerNormQueue lnq;
    };

    Queues * qs;
};

using BlockShape = cutlass::gemm::GemmShape<MgnFullMlp::mblk, 128, 128>;
using LayerNormBlock = LayerNormShape<MgnFullMlp::mblk, 128>;


const size_t max_smem = std::max({
    sizeof(typename PipeGemm<BlockShape>::SmemBuffers),
    sizeof(typename PipeGemmBias<BlockShape>::SmemBuffers),
    sizeof(typename PipeGemmBiasRelu<BlockShape>::SmemBuffers),
    sizeof(LayerNormSmemBuffers<128, num_warps>)
});


__device__ void mlp0_sm0(MgnFullMlp& prob, int row) {
    if (threadIdx.y >= PipeGemm<BlockShape>::num_warps) return;
    const int num_iters = prob.m / MgnFullMlp::mblk / MgnFullMlp::n_rows;

    MemoryReader ir(
        &prob.in[row * num_iters * MgnFullMlp::mblk * MgnFullMlp::d * 3 + 0],
        MgnFullMlp::mblk * MgnFullMlp::d * 3,
        MgnFullMlp::d * 3);

    NullReader ar;
    QueueWriter ow(prob.qs[row].q01);

    pipe_gemm<BlockShape>(
        {&prob.w1[0], MgnFullMlp::d},
        ir,
        ar,
        ow,
        num_iters);
}


__device__ void mlp0_sm1(MgnFullMlp& prob, int row) {
    if (threadIdx.y >= PipeGemm<BlockShape>::num_warps) return;
    const int num_iters = prob.m / MgnFullMlp::mblk / MgnFullMlp::n_rows;

    MemoryReader ir(
        &prob.in[row * num_iters * MgnFullMlp::mblk * MgnFullMlp::d * 3 + 128],
        MgnFullMlp::mblk * MgnFullMlp::d * 3,
        MgnFullMlp::d * 3);

    QueueReader ar(prob.qs[row].q01);
    QueueWriter ow(prob.qs[row].q12);

    pipe_gemm<BlockShape>(
        {&prob.w1[128 * MgnFullMlp::d], MgnFullMlp::d},
        ir,
        ar,
        ow,
        num_iters);
}

__device__ void mlp0_sm2(MgnFullMlp& prob, int row) {
    if (threadIdx.y >= PipeGemm<BlockShape>::num_warps) return;
    const int num_iters = prob.m / MgnFullMlp::mblk / MgnFullMlp::n_rows;

    MemoryReader ir(
        &prob.in[row * num_iters * MgnFullMlp::mblk * MgnFullMlp::d * 3 + 256],
        MgnFullMlp::mblk * MgnFullMlp::d * 3,
        MgnFullMlp::d * 3);

    QueueReader ar(prob.qs[row].q12);
    QueueWriter ow(prob.qs[row].q23);

    pipe_gemm_bias_relu<BlockShape>(
        {&prob.w1[256 * MgnFullMlp::d], MgnFullMlp::d},
        {&prob.b1[0], 0},
        ir,
        ar,
        ow,
        num_iters);
}

__device__ void mlp1_sm0(MgnFullMlp& prob, int row) {
    if (threadIdx.y >= PipeGemm<BlockShape>::num_warps) return;
    const int num_iters = prob.m / MgnFullMlp::mblk / MgnFullMlp::n_rows;

    QueueReader ir(prob.qs[row].q23);
    NullReader ar;
    QueueWriter ow(prob.qs[row].q34);

    pipe_gemm_bias_relu<BlockShape>(
        {&prob.w2[0], MgnFullMlp::d},
        {&prob.b2[0], 0},
        ir,
        ar,
        ow,
        num_iters);
}

__device__ void mlp2_sm0(MgnFullMlp& prob, int row) {
    if (threadIdx.y >= PipeGemm<BlockShape>::num_warps) return;
    const int num_iters = prob.m / MgnFullMlp::mblk / MgnFullMlp::n_rows;

    QueueReader ir(prob.qs[row].q34);
    NullReader ar;
    // QueueWriter ow(prob.qs[row].lnq);

    MemoryWriter ow(
        &prob.out[row * num_iters * MgnFullMlp::mblk * MgnFullMlp::d],
        MgnFullMlp::mblk * MgnFullMlp::d,
        MgnFullMlp::d);

    pipe_gemm_bias<BlockShape>(
        {&prob.w3[0], MgnFullMlp::d},
        {&prob.b3[0], 0},
        ir,
        ar,
        ow,
        num_iters);
}

__device__ void ln_sm(MgnFullMlp& prob, int row, int ln) {
    const int num_iters_per_row = prob.m / MgnFullMlp::mblk / MgnFullMlp::n_rows;
    const int num_iters =
        num_iters_per_row / MgnFullMlp::n_ln_cols +
        (ln < num_iters_per_row % MgnFullMlp::n_ln_cols ? 1 : 0);


    SplitQueueReader ir(prob.qs[row].lnq, ln, MgnFullMlp::n_ln_cols);
    NullReader ar;
    MemoryWriter ow(
        &prob.out[(row * num_iters_per_row + ln) * MgnFullMlp::mblk * MgnFullMlp::d],
        MgnFullMlp::n_ln_cols * MgnFullMlp::mblk * MgnFullMlp::d,
        MgnFullMlp::d);

    pipe_layer_norm<num_warps, LayerNormBlock>(
        {&prob.gamma[0], 0},
        {&prob.beta[0], 0},
        ir,
        ow,
        num_iters);
}

__global__ void fullmlp_device(
    int m,
    half * x,     // [M, 384]
    half * w1,    // [384, 128]
    half * b1,    // [128]
    half * w2,    // [128, 128]
    half * b2,    // [128]
    half * w3,    // [128, 128]
    half * b3,    // [128]
    half * gamma, // [128]
    half * beta,  // [128]
    half * out,   // [M, 128]
    void * qs
) {
    int pipe_col = blockIdx.x;
    int pipe_row = blockIdx.y;

    MgnFullMlp prob = {
        .m = m,
        .in = x,
        .w1 = w1,
        .b1 = b1,
        .w2 = w2,
        .b2 = b2,
        .w3 = w3,
        .b3 = b3,
        .gamma = gamma,
        .beta = beta,
        .out = out,
        .qs = (typename MgnFullMlp::Queues *)qs
    };

    switch (pipe_col) {
        case 0: mlp0_sm0(prob, pipe_row); break;
        case 1: mlp0_sm1(prob, pipe_row); break;
        case 2: mlp0_sm2(prob, pipe_row); break;
        case 3: mlp1_sm0(prob, pipe_row); break;
        case 4: mlp2_sm0(prob, pipe_row); break;
        default:
            pipe_col -= MgnFullMlp::n_mlp_cols;

            if (pipe_col < MgnFullMlp::n_ln_cols) {
                ln_sm(prob, pipe_row, pipe_col);
            }

            return;
    }
}


inline typename MgnFullMlp::Queues * global_queue_space() {
    static typename MgnFullMlp::Queues * qs_dev = nullptr;

    if (qs_dev != nullptr) return qs_dev;

    cudaErrCheck(hipMalloc(&qs_dev, MgnFullMlp::n_rows * sizeof(*qs_dev)));
    cudaErrCheck(hipMemset(qs_dev, 0, MgnFullMlp::n_rows * sizeof(*qs_dev)));

    pin_memory(qs_dev, MgnFullMlp::n_rows * sizeof(*qs_dev));

    return qs_dev;
}

inline void configure_smem_once() {
    static bool configured = false;
    if (configured) return;
    configure_smem((const void *)fullmlp_device, max_smem);
    configured = true;
}


void mgn_fullmlp_out(
    at::Tensor x,     // [M, 384]
    at::Tensor w1,    // [384, 128]
    at::Tensor b1,    // [128]
    at::Tensor w2,    // [128, 128]
    at::Tensor b2,    // [128]
    at::Tensor w3,    // [128, 128]
    at::Tensor b3,    // [128]
    at::Tensor gamma, // [128]
    at::Tensor beta,  // [128]
    at::Tensor out    // [M, 128]
) {
    CHECK_INPUT(x);
    CHECK_INPUT(w1);
    CHECK_INPUT(b1);
    CHECK_INPUT(w2);
    CHECK_INPUT(b2);
    CHECK_INPUT(w3);
    CHECK_INPUT(b3);

    assert(x.size(1) == 384);
    assert(w1.size(0) == 384 && w1.size(1) == 128 && b1.size(0) == 128);
    assert(w2.size(0) == 128 && w2.size(1) == 128 && b2.size(0) == 128);
    assert(w3.size(0) == 128 && w3.size(1) == 128 && b3.size(0) == 128);
    assert(gamma.size(0) == 128 && beta.size(0) == 128);

    dim3 grid(MgnFullMlp::n_mlp_cols, MgnFullMlp::n_rows);
    dim3 block(32, num_warps);

    configure_smem_once();

    fullmlp_device<<<grid, block, max_smem>>>(
        x.size(0),
        (half *)x.data_ptr<at::Half>(),
        (half *)w1.data_ptr<at::Half>(),
        (half *)b1.data_ptr<at::Half>(),
        (half *)w2.data_ptr<at::Half>(),
        (half *)b2.data_ptr<at::Half>(),
        (half *)w3.data_ptr<at::Half>(),
        (half *)b3.data_ptr<at::Half>(),
        (half *)gamma.data_ptr<at::Half>(),
        (half *)beta.data_ptr<at::Half>(),
        (half *)out.data_ptr<at::Half>(),
        global_queue_space()
    );
}


at::Tensor mgn_fullmlp(
    at::Tensor x,     // [M, 384]
    at::Tensor w1,    // [384, 128]
    at::Tensor b1,    // [128]
    at::Tensor w2,    // [128, 128]
    at::Tensor b2,    // [128]
    at::Tensor w3,    // [128, 128]
    at::Tensor b3,    // [128]
    at::Tensor gamma, // [128]
    at::Tensor beta   // [128]
) {
    at::Tensor out = at::zeros({x.size(0), 128}, x.options());
    mgn_fullmlp_out(x, w1, b1, w2, b2, w3, b3, gamma, beta, out);
    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("mgn_fullmlp", &mgn_fullmlp, "mgn_fullmlp");
    m.def("mgn_fullmlp_out", &mgn_fullmlp_out, "mgn_fullmlp_out");
}

