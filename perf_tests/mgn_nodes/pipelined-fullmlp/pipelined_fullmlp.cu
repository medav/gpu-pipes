#include "hip/hip_runtime.h"
#include "mgn_node_pipe.cuh"
#include "pipes.cuh"
#include "pipegemm.cuh"
#include "pipebiasaddrelu.cuh"
#include "utils.cuh"

const size_t max_smem = sizeof(SmemBuffers<MgnNodeMlp::mblk, 128, 128>);

__device__ void mlp0_gemm0(void * smem, MgnNodeMlp *prob, size_t row) {
    using Input = MemoryReader;
    using Accum = NullReader;
    using Output = QueueWriter<MgnNodeMlp::Queue>;

    const size_t mblk = MgnNodeMlp::mblk;
    const size_t num_iters = prob->mi / MgnNodeMlp::mblk;

    Input input(&prob->in[0][row * prob->mi][0], mblk * prob->d);
    Accum accum;
    Output output(prob->qs.qs[row][0]);

    for (size_t i = 0; i < MgnNodeMlp::ni; i++) {
        input.reset();
        gemmpipe<
            cutlass::gemm::GemmShape<mblk, 128, 128>,
            Input,
            Accum,
            Output
        >(smem, &prob->w1[0][0][0], input, accum, output, num_iters);
    }
}

__device__ void mlp0_gemm1(void * smem, MgnNodeMlp *prob, size_t row) {
    using Input = MemoryReader;
    using Accum = QueueReader<MgnNodeMlp::Queue>;
    using Output = QueueWriter<MgnNodeMlp::Queue>;

    const size_t mblk = MgnNodeMlp::mblk;
    const size_t num_iters = prob->mi / MgnNodeMlp::mblk;

    Input input(&prob->in[1][row * prob->mi][0], mblk * prob->d);
    Accum accum(prob->qs.qs[row][0]);
    Output output(prob->qs.qs[row][1]);

    for (size_t i = 0; i < MgnNodeMlp::ni; i++) {
        input.reset();
        gemmpipe<
            cutlass::gemm::GemmShape<mblk, 128, 128>,
            Input,
            Accum,
            Output
        >(smem, &prob->w1[1][0][0], input, accum, output, num_iters);
    }
}

__device__ void mlp0_gemm2(void * smem, MgnNodeMlp *prob, size_t row) {
    using Input = MemoryReader;
    using Accum = QueueReader<MgnNodeMlp::Queue>;
    using Output = QueueWriter<MgnNodeMlp::Queue>;

    const size_t mblk = MgnNodeMlp::mblk;
    const size_t num_iters = prob->mi / MgnNodeMlp::mblk;

    Input input(&prob->in[2][row * prob->mi][0], mblk * prob->d);
    Accum accum(prob->qs.qs[row][1]);
    Output output(prob->qs.qs[row][2]);


    for (size_t i = 0; i < MgnNodeMlp::ni; i++) {
        input.reset();
        gemmpipe<
            cutlass::gemm::GemmShape<mblk, 128, 128>,
            Input,
            Accum,
            Output
        >(smem, &prob->w1[2][0][0], input, accum, output, num_iters);
    }
}

__device__ void mlp0_bias_relu(void * smem, MgnNodeMlp *prob, size_t row) {
    using Input = QueueReader<MgnNodeMlp::Queue>;
    using Accum = NullReader;
    using Output = QueueWriter<MgnNodeMlp::Queue>;

    const size_t mblk = MgnNodeMlp::mblk;
    const size_t num_iters = prob->mi / MgnNodeMlp::mblk;

    Input input(prob->qs.qs[row][2]);
    Accum accum;
    Output output(prob->qs.qs[row][3]);

    for (size_t i = 0; i < MgnNodeMlp::ni; i++) {
        biasaddrelupipe<
            cutlass::gemm::GemmShape<mblk, 128, 128>,
            Input,
            Accum,
            Output
        >(smem, &prob->b1[0], input, accum, output, num_iters);
    }
}

__device__ void mlp1_gemm0(void * smem, MgnNodeMlp *prob, size_t row) {
    using Input = QueueReader<MgnNodeMlp::Queue>;
    using Accum = NullReader;
    using Output = QueueWriter<MgnNodeMlp::Queue>;

    const size_t mblk = MgnNodeMlp::mblk;
    const size_t num_iters = prob->mi / MgnNodeMlp::mblk;

    Input input(prob->qs.qs[row][3]);
    Accum accum;
    Output output(prob->qs.qs[row][4]);

    for (size_t i = 0; i < MgnNodeMlp::ni; i++) {
        gemmpipe<
            cutlass::gemm::GemmShape<mblk, 128, 128>,
            Input,
            Accum,
            Output
        >(smem, &prob->w2[0][0], input, accum, output, num_iters);
    }
}

__device__ void mlp1_bias_relu(void * smem, MgnNodeMlp *prob, size_t row) {
    using Input = QueueReader<MgnNodeMlp::Queue>;
    using Accum = NullReader;
    using Output = QueueWriter<MgnNodeMlp::Queue>;

    const size_t mblk = MgnNodeMlp::mblk;
    const size_t num_iters = prob->mi / MgnNodeMlp::mblk;

    Input input(prob->qs.qs[row][4]);
    Accum accum;
    Output output(prob->qs.qs[row][5]);

    for (size_t i = 0; i < MgnNodeMlp::ni; i++) {
        biasaddrelupipe<
            cutlass::gemm::GemmShape<mblk, 128, 128>,
            Input,
            Accum,
            Output
        >(smem, &prob->b2[0], input, accum, output, num_iters);
    }
}

__device__ void mlp2_gemm0(void * smem, MgnNodeMlp *prob, size_t row) {
    using Input = QueueReader<MgnNodeMlp::Queue>;
    using Accum = NullReader;
    using Output = QueueWriter<MgnNodeMlp::Queue>;

    const size_t mblk = MgnNodeMlp::mblk;
    const size_t num_iters = prob->mi / MgnNodeMlp::mblk;

    Input input(prob->qs.qs[row][5]);
    Accum accum;
    Output output(prob->qs.qs[row][6]);


    for (size_t i = 0; i < MgnNodeMlp::ni; i++) {
        gemmpipe<
            cutlass::gemm::GemmShape<mblk, 128, 128>,
            Input,
            Accum,
            Output
        >(smem, &prob->w3[0][0], input, accum, output, num_iters);
    }
}

__device__ void mlp2_bias_relu(void * smem, MgnNodeMlp *prob, size_t row) {
    using Input = QueueReader<MgnNodeMlp::Queue>;
    using Accum = NullReader;
    // using Output = QueueWriter<MgnNodeMlp::Queue>;
    using Output = MemoryWriter;

    const size_t mblk = MgnNodeMlp::mblk;
    const size_t num_iters = prob->mi / MgnNodeMlp::mblk;

    Input input(prob->qs.qs[row][6]);
    Accum accum;
    // Output output(prob->qs.qs[row][7]);
    Output output(&prob->out[row * prob->mo][0], mblk * prob->d);

    for (size_t i = 0; i < MgnNodeMlp::ni; i++) {
        output.reset();
        biasaddrelupipe<
            cutlass::gemm::GemmShape<mblk, 128, 128>,
            Input,
            Accum,
            Output
        >(smem, &prob->b2[0], input, accum, output, num_iters);
    }
}

template<typename QT>
__device__ void consume_dummy(QT& q, size_t num_iters) {
    QueueReader<QT> r(q);

    r.reset();

    for (size_t i = 0; i < num_iters; i++) {
        r.read_acquire();
        r.read_release();
    }
}


__global__ void kernel(MgnNodeMlp * prob) {
    void * smem = nullptr;
    size_t pipe_col = blockIdx.x;
    size_t pipe_row = blockIdx.y;

    switch (pipe_col) {
        case 0: mlp0_gemm0(smem, prob, pipe_row); break;
        case 1: mlp0_gemm1(smem, prob, pipe_row); break;
        // case 2: consume_dummy(prob->qs.qs[pipe_row][1], prob->mi / MgnNodeMlp::mblk); break;
        case 2: mlp0_gemm2(smem, prob, pipe_row); break;
        case 3: mlp0_bias_relu(smem, prob, pipe_row); break;
        // case 4: consume_dummy(prob->qs.qs[pipe_row][3], prob->mi / MgnNodeMlp::mblk); break;
        case 4: mlp1_gemm0(smem, prob, pipe_row); break;
        case 5: mlp1_bias_relu(smem, prob, pipe_row); break;
        case 6: mlp2_gemm0(smem, prob, pipe_row); break;
        case 7: mlp2_bias_relu(smem, prob, pipe_row); break;
        // case 8: consume_dummy(prob->qs.qs[pipe_row][7], prob->mi / MgnNodeMlp::mblk); break;
        default: return;
    }
}

int main() {

    cudaErrCheck(hipDeviceSetCacheConfig(hipFuncCachePreferShared));
    cudaErrCheck(hipFuncSetAttribute(reinterpret_cast<const void*>(
        kernel),
        hipFuncAttributePreferredSharedMemoryCarveout,
        cudaSharedmemCarveoutMaxShared));

    cudaErrCheck(hipFuncSetAttribute(reinterpret_cast<const void*>(
        kernel), hipFuncAttributeMaxDynamicSharedMemorySize, max_smem));

    MgnNodeMlp * prob;
    cudaErrCheck(hipMalloc(&prob, sizeof(MgnNodeMlp) + 128));
    // Align prob
    // prob = (MgnNodeMlp*)(((size_t)prob + 0x3F) & ~0x3F);

    // Print address of prob
    printf("prob: %p\n", prob);


    size_t tot_pipe_bytes = sizeof(MgnNodeMlp::Queues);

    printf("Total pipe bytes: %lu\n", tot_pipe_bytes);
    printf("Total pipe bytes: %lu KB\n", tot_pipe_bytes / 1024);

    printf("Init...\n");
    init_prob<<<1, 128>>>(prob);
    cudaErrCheck(hipDeviceSynchronize());

    hipLaunchAttributeValue attribute;
    auto& window = attribute.accessPolicyWindow;
    window.base_ptr = &prob->qs;
    window.num_bytes = sizeof(MgnNodeMlp::Queues);
    window.hitRatio = 1.0;
    window.hitProp = hipAccessPropertyPersisting;
    window.missProp = hipAccessPropertyStreaming;

    hipStreamSetAttribute(
        hipStreamDefault,
        hipLaunchAttributeAccessPolicyWindow,
        &attribute
    );

    dim3 grid(9, MgnNodeMlp::mo);
    // dim3 grid(5, 1);
    dim3 block(32, num_warps);

    const size_t tot_loop_iters = MgnNodeMlp::ni * MgnNodeMlp::mi / MgnNodeMlp::mblk;
    printf("Total loop iters: %lu\n", tot_loop_iters);

    printf("SMEM: %lu\n", max_smem);
    printf("# Warps: %lu\n", num_warps);

    printf("Running...\n");
    float time_ms = cuda_time_kernel_ms(
        [&]() {
            kernel<<<grid, block, max_smem>>>(prob);
        }
    );

    printf("Total time: %f ms\n", time_ms);
    printf("Avg. loop iter time: %f ms\n", time_ms / tot_loop_iters);

    float flops_v1 =
        2.0f * MgnNodeMlp::m * (3 * MgnNodeMlp::d) * MgnNodeMlp::d +
        2.0f * MgnNodeMlp::m * MgnNodeMlp::d * MgnNodeMlp::d +
        2.0f * MgnNodeMlp::m * MgnNodeMlp::d * MgnNodeMlp::d;
    float gflops_v1 = MgnNodeMlp::ni * flops_v1 / (time_ms * 1e6);
    printf("+ GFLOPS: %f\n", gflops_v1);

    return 0;
}
