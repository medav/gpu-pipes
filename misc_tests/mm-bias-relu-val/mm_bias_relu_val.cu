#include "hip/hip_runtime.h"
#include "cutlass/cutlass.h"
#include "cutlass/gemm/gemm.h"
#include "cutlass/gemm/warp/mma.h"
#include "cutlass/gemm/warp/default_mma_tensor_op.h"
#include "cutlass/gemm/threadblock/default_mma.h"
#include "cutlass/gemm/threadblock/default_mma_core.h"
#include "cutlass/gemm/threadblock/mma_pipelined.h"
#include "cutlass/epilogue/collective/default_epilogue.hpp"
#include "cutlass/epilogue/thread/linear_combination.h"
#include "cutlass/epilogue/warp/fragment_iterator_tensor_op.h"
#include "cutlass/epilogue/warp/tile_iterator_tensor_op.h"
#include "cutlass/epilogue/threadblock/default_epilogue_tensor_op.h"

#include "cutlass/arch/wmma.h"
#include "cutlass/numeric_types.h"
#include "cutlass/arch/arch.h"
#include "cutlass/arch/mma.h"
#include "cutlass/layout/matrix.h"
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/default_gemm_universal.h"

#include "mpmcq.cuh"
#include "pipes.cuh"
#include "utils.cuh"

#include "pipe_gemm.cuh"
#include "pipe_gemm_bias.cuh"
#include "pipe_gemm_bias_relu.cuh"

#define MM 128
#define NN 128
#define KK 128


struct Tensor {
    const size_t R;
    const size_t C;

    float * host_ptr;
    half * dev_ptr;

    Tensor(size_t R, size_t C) : R(R), C(C) {
        host_ptr = new float[R * C];
        hipMalloc(&dev_ptr, R * C * sizeof(half));
    }

    ~Tensor() {
        delete[] host_ptr;
        hipFree(dev_ptr);
    }

    void rand_fill() {
        for (size_t i = 0; i < R * C; i++) {
            host_ptr[i] = 2.0f * ((float)rand() / (float)RAND_MAX - 0.5f);
            // host_ptr[i] = (float)rand() / RAND_MAX;
        }
    }

    void fill(float val) {
        for (size_t i = 0; i < R * C; i++) {
            host_ptr[i] = val;
        }
    }

    void to_dev() {
        float * tmp_dev;
        hipMalloc(&tmp_dev, R * C * sizeof(float));
        hipMemcpy(tmp_dev, host_ptr, R * C * sizeof(float), hipMemcpyHostToDevice);
        float_to_half<<<CLD(R * C, 128), 128>>>(dev_ptr, tmp_dev, R * C);
        hipFree(tmp_dev);
    }

    void to_host() {
        float * tmp_dev;
        hipMalloc(&tmp_dev, R * C * sizeof(float));
        half_to_float<<<CLD(R * C, 128), 128>>>(tmp_dev, dev_ptr, R * C);
        hipMemcpy(host_ptr, tmp_dev, R * C * sizeof(float), hipMemcpyDeviceToHost);
        hipFree(tmp_dev);
    }

    void print() {
        for (size_t r = 0; r < R; r++) {
            for (size_t c = 0; c < C; c++) {
                printf("%.2f ", host_ptr[r * C + c]);
            }
            printf("\n");
        }
    }
};

__global__ void ref_gemm(half * x, half * w, half * out) {
    int m = blockIdx.x * blockDim.x + threadIdx.x;
    int n = blockIdx.y * blockDim.y + threadIdx.y;

    if (m >= MM || n >= NN) return;

    float sum = 0.0f;
    for (int k = 0; k < KK; k++) {
        sum += (float)x[m * KK + k] * (float)w[k * NN + n];
    }
    out[m * NN + n] = (half)sum;
}

__global__ void ref_gemm_bias(half * x, half * w, half * b, half * out) {
    int m = blockIdx.x * blockDim.x + threadIdx.x;
    int n = blockIdx.y * blockDim.y + threadIdx.y;

    if (m >= MM || n >= NN) return;

    float sum = 0.0f;
    for (int k = 0; k < KK; k++) {
        sum += (float)x[m * KK + k] * (float)w[k * NN + n];
    }
    sum += (float)b[n];
    out[m * NN + n] = (half)sum;
}

__global__ void ref_gemm_bias_relu(half * x, half * w, half * b, half * out) {
    int m = blockIdx.x * blockDim.x + threadIdx.x;
    int n = blockIdx.y * blockDim.y + threadIdx.y;

    if (m >= MM || n >= NN) return;

    float sum = 0.0f;
    for (int k = 0; k < KK; k++) {
        sum += (float)x[m * KK + k] * (float)w[k * NN + n];
    }
    sum += (float)b[n];
    out[m * NN + n] = sum > 0.0f ? (half)sum : (half)0.0f;
}

float rel_err(float a, float b, float eps = 1e-6f) {
    return fabs(a - b) / ((a + b) / 2.0f + eps);
}

bool isclose(float a, float b, float rtol = 0.05) {
    return rel_err(a, b) < rtol;
}


void compare(Tensor& ref, Tensor& act) {
    for (size_t r = 0; r < ref.R; r++) {
        for (size_t c = 0; c < ref.C; c++) {
            float ref_val = ref.host_ptr[r * ref.C + c];
            float act_val = act.host_ptr[r * act.C + c];
            if (!isclose(ref_val, act_val)) {
                printf("Mismatch at %zu, %zu: %f != %f\n", r, c, ref_val, act_val);
            }
        }
    }
}

float l2(Tensor& a, Tensor& b) {
    float sum_sq = 0.0f;
    for (size_t r = 0; r < a.R; r++) {
        for (size_t c = 0; c < a.C; c++) {
            float ax = a.host_ptr[r * a.C + c];
            float bx = b.host_ptr[r * a.C + c];

            sum_sq += (ax - bx) * (ax - bx);
        }
    }

    return sqrt(sum_sq);
}

void configure_smem(const void * func, const size_t smem) {
    cudaErrCheck(hipDeviceSetCacheConfig(hipFuncCachePreferShared));
    cudaErrCheck(hipFuncSetAttribute(reinterpret_cast<const void*>(
        func),
        hipFuncAttributePreferredSharedMemoryCarveout,
        cudaSharedmemCarveoutMaxShared));

    cudaErrCheck(hipFuncSetAttribute(reinterpret_cast<const void*>(
        func),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        smem));
}

__global__ void test_pipe_gemm_kernel(half * x, half * w, half * out) {
    using Input = MemoryReader;
    using Accum = NullReader;
    using Output = MemoryWriter;

    Input ir(x, 0);
    Accum ar;
    Output ow(out, 0);

    pipe_gemm<
        cutlass::gemm::GemmShape<MM, NN, KK>,
        Input,
        Accum,
        Output
    >(w, ir, ar, ow, 1);
}


void test_pipe_gemm() {
    using Types = PipeGemm<cutlass::gemm::GemmShape<MM, NN, KK>>;
    using SmemBuffers = Types::SmemBuffers;

    printf("==== test_pipe_gemm ====\n");
    Tensor x(128, 128);
    Tensor w(128, 128);
    Tensor out(128, 128);
    Tensor ref(128, 128);

    x.rand_fill();
    w.rand_fill();

    x.to_dev();
    w.to_dev();

    const size_t smem = sizeof(SmemBuffers);
    configure_smem((const void *)test_pipe_gemm_kernel, smem);

    cuda_time_kernel_ms([&] () {
        dim3 block(32, Types::num_warps);
        test_pipe_gemm_kernel<<<1, block, smem>>>(
            x.dev_ptr, w.dev_ptr, out.dev_ptr);
    });

    cuda_time_kernel_ms([&] () {
        dim3 block(32, 32);
        dim3 grid(4, 4);
        ref_gemm<<<grid, block>>>(x.dev_ptr, w.dev_ptr, ref.dev_ptr);
    });

    out.to_host();
    ref.to_host();
    compare(ref, out);
    printf("L2 error: %.6f\n", l2(ref, out));
    printf("\n");
}

__global__ void test_pipe_gemm_bias_kernel(half * x, half * w, half * b, half * out) {
    using Input = MemoryReader;
    using Accum = NullReader;
    using Output = MemoryWriter;

    Input ir(x, 0);
    Accum ar;
    Output ow(out, 0);

    pipe_gemm_bias<
        cutlass::gemm::GemmShape<MM, NN, KK>,
        Input,
        Accum,
        Output
    >(w, b, ir, ar, ow, 1);
}


void test_pipe_gemm_bias() {
    using Types = PipeGemmBias<cutlass::gemm::GemmShape<MM, NN, KK>>;
    using SmemBuffers = Types::SmemBuffers;

    printf("==== test_pipe_gemm_bias ====\n");
    Tensor x(128, 128);
    Tensor w(128, 128);
    Tensor b(1, 128);
    Tensor out(128, 128);
    Tensor ref(128, 128);

    x.rand_fill();
    w.rand_fill();
    b.fill(1.0f);

    x.to_dev();
    w.to_dev();
    b.to_dev();

    const size_t smem = sizeof(SmemBuffers);
    configure_smem((const void *)test_pipe_gemm_bias_kernel, smem);

    cuda_time_kernel_ms([&] () {
        dim3 block(32, Types::num_warps);
        test_pipe_gemm_bias_kernel<<<1, block, smem>>>(
            x.dev_ptr, w.dev_ptr, b.dev_ptr, out.dev_ptr);
    });

    cuda_time_kernel_ms([&] () {
        dim3 block(32, 32);
        dim3 grid(4, 4);
        ref_gemm_bias<<<grid, block>>>(
            x.dev_ptr, w.dev_ptr, b.dev_ptr, ref.dev_ptr);
    });

    out.to_host();
    ref.to_host();
    compare(ref, out);
    printf("L2 error: %.6f\n", l2(ref, out));
    printf("\n");
}

__global__ void test_pipe_gemm_bias_relu_kernel(half * x, half * w, half * b, half * out) {
    using Input = MemoryReader;
    using Accum = NullReader;
    using Output = MemoryWriter;

    Input ir(x, 0);
    Accum ar;
    Output ow(out, 0);

    pipe_gemm_bias_relu<
        cutlass::gemm::GemmShape<MM, NN, KK>,
        Input,
        Accum,
        Output
    >(w, b, ir, ar, ow, 1);
}


void test_pipe_gemm_bias_relu() {
    using Types = PipeGemmBiasRelu<cutlass::gemm::GemmShape<MM, NN, KK>>;
    using SmemBuffers = Types::SmemBuffers;

    printf("==== test_pipe_gemm_bias_relu ====\n");
    Tensor x(128, 128);
    Tensor w(128, 128);
    Tensor b(1, 128);
    Tensor out(128, 128);
    Tensor ref(128, 128);

    x.rand_fill();
    w.rand_fill();
    b.fill(1.0f);

    x.to_dev();
    w.to_dev();
    b.to_dev();

    dim3 block(32, Types::num_warps);
    const size_t smem = sizeof(SmemBuffers);
    configure_smem((const void *)test_pipe_gemm_bias_relu_kernel, smem);

    cuda_time_kernel_ms([&] () {
        test_pipe_gemm_bias_relu_kernel<<<1, block, smem>>>(
            x.dev_ptr, w.dev_ptr, b.dev_ptr, out.dev_ptr);
    });

    cuda_time_kernel_ms([&] () {
        dim3 block(32, 32);
        dim3 grid(4, 4);
        ref_gemm_bias_relu<<<grid, block>>>(
            x.dev_ptr, w.dev_ptr, b.dev_ptr, ref.dev_ptr);
    });

    out.to_host();
    ref.to_host();
    compare(ref, out);
    printf("L2 error: %.6f\n", l2(ref, out));
    printf("\n");
}

int main(){
    srand(time(0));
    test_pipe_gemm();
    test_pipe_gemm_bias();
    test_pipe_gemm_bias_relu();
    return 0;
}
