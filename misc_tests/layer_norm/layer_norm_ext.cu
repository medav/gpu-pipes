

#include <torch/extension.h>
#include <ATen/ATen.h>

#include <hip/hip_fp16.h>
#include "layer_norm.cuh"
#include "utils.cuh"

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)


at::Tensor layer_norm_128(
    at::Tensor x,
    at::Tensor gamma,
    at::Tensor beta
) {
    CHECK_INPUT(x);
    CHECK_INPUT(gamma);
    CHECK_INPUT(beta);

    const int MBLK = 64;

    at::Tensor out = at::empty_like(x);

    assert(x.size(1) == 128);

    dim3 grid(x.size(0) / MBLK);
    dim3 block(32, 4);

    device_layer_norm<128><<<grid, block>>>(
        (half *)x.data_ptr<at::Half>(),
        (half *)gamma.data_ptr<at::Half>(),
        (half *)beta.data_ptr<at::Half>(),
        (half *)out.data_ptr<at::Half>(),
        MBLK
    );

    return out;
}

float bench_layer_norm_128(
    at::Tensor x,
    at::Tensor gamma,
    at::Tensor beta,
    int MBLK,
    int NW,
    int ni
) {
    CHECK_INPUT(x);
    CHECK_INPUT(gamma);
    CHECK_INPUT(beta);

    at::Tensor out = at::empty_like(x);

    assert(x.size(1) == 128);

    dim3 grid(x.size(0) / MBLK);
    dim3 block(32, NW);

    float time_ms = cuda_time_kernel_ms([&]() {
        for (int i = 0; i < ni; i++) {
            device_layer_norm<128><<<grid, block>>>(
                (half *)x.data_ptr<at::Half>(),
                (half *)gamma.data_ptr<at::Half>(),
                (half *)beta.data_ptr<at::Half>(),
                (half *)out.data_ptr<at::Half>(),
                MBLK
            );
        }
    });

    return time_ms;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("layer_norm_128", &layer_norm_128, "layer_norm_128");
    m.def("bench_layer_norm_128", &bench_layer_norm_128, "bench_layer_norm_128");
}

