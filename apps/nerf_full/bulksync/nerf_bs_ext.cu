

#include <torch/extension.h>
#include <ATen/ATen.h>

#include <hip/hip_fp16.h>

#include "nerf_bs.cuh"
#include "bulksync_gemm.cuh"
#include "utils.cuh"

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)


void nerf_out(
    at::Tensor x,
    at::Tensor d,
    at::Tensor w1, at::Tensor b1,
    at::Tensor w2, at::Tensor b2,
    at::Tensor w3, at::Tensor b3,
    at::Tensor w4, at::Tensor b4,
    at::Tensor w5, at::Tensor b5,
    at::Tensor w6, at::Tensor b6,
    at::Tensor w7, at::Tensor b7,
    at::Tensor w8, at::Tensor b8,
    at::Tensor w9, at::Tensor b9,
    at::Tensor w10, at::Tensor b10,
    at::Tensor w11, at::Tensor b11,
    at::Tensor w12, at::Tensor b12,
    at::Tensor t1,
    at::Tensor t2,
    at::Tensor x_out,
    at::Tensor r_out
) {
    CHECK_INPUT(x);
    CHECK_INPUT(w1); CHECK_INPUT(b1);
    CHECK_INPUT(w2); CHECK_INPUT(b2);
    CHECK_INPUT(w3); CHECK_INPUT(b3);
    CHECK_INPUT(w4); CHECK_INPUT(b4);
    CHECK_INPUT(w5); CHECK_INPUT(b5);
    CHECK_INPUT(w6); CHECK_INPUT(b6);
    CHECK_INPUT(w7); CHECK_INPUT(b7);
    CHECK_INPUT(w8); CHECK_INPUT(b8);
    CHECK_INPUT(w9); CHECK_INPUT(b9);
    CHECK_INPUT(w10); CHECK_INPUT(b10);
    CHECK_INPUT(w11); CHECK_INPUT(b11);
    CHECK_INPUT(w12); CHECK_INPUT(b12);

    assert(x.size(0) == MM);
    assert(x.size(1) == 64);

    assert(d.size(0) == MM);
    assert(d.size(1) == 32);

    assert(w1.size(0) == 64 && w1.size(1) == 256 && b1.size(0) == 256);
    assert(w2.size(0) == 256 && w2.size(1) == 256 && b2.size(0) == 256);
    assert(w3.size(0) == 256 && w3.size(1) == 256 && b3.size(0) == 256);
    assert(w4.size(0) == 256 && w4.size(1) == 256 && b4.size(0) == 256);
    assert(w5.size(0) == 256 && w5.size(1) == 256 && b5.size(0) == 256);
    assert(w6.size(0) == 320 && w6.size(1) == 256 && b6.size(0) == 256);
    assert(w7.size(0) == 256 && w7.size(1) == 256 && b7.size(0) == 256);
    assert(w8.size(0) == 256 && w8.size(1) == 256 && b8.size(0) == 256);
    assert(w9.size(0) == 256 && w9.size(1) == 16 && b9.size(0) == 16);
    assert(w10.size(0) == 256 && w10.size(1) == 256 && b10.size(0) == 256);
    assert(w11.size(0) == 288 && w11.size(1) == 128 && b11.size(0) == 128);
    assert(w12.size(0) == 128 && w12.size(1) == 16 && b12.size(0) == 16);

    nerf_bs<MM>(
        (half *)x.data_ptr<at::Half>(),
        (half *)d.data_ptr<at::Half>(),
        (half *)w1.data_ptr<at::Half>(), (half *)b1.data_ptr<at::Half>(),
        (half *)w2.data_ptr<at::Half>(), (half *)b2.data_ptr<at::Half>(),
        (half *)w3.data_ptr<at::Half>(), (half *)b3.data_ptr<at::Half>(),
        (half *)w4.data_ptr<at::Half>(), (half *)b4.data_ptr<at::Half>(),
        (half *)w5.data_ptr<at::Half>(), (half *)b5.data_ptr<at::Half>(),
        (half *)w6.data_ptr<at::Half>(), (half *)b6.data_ptr<at::Half>(),
        (half *)w7.data_ptr<at::Half>(), (half *)b7.data_ptr<at::Half>(),
        (half *)w8.data_ptr<at::Half>(), (half *)b8.data_ptr<at::Half>(),
        (half *)w9.data_ptr<at::Half>(), (half *)b9.data_ptr<at::Half>(),
        (half *)w10.data_ptr<at::Half>(), (half *)b10.data_ptr<at::Half>(),
        (half *)w11.data_ptr<at::Half>(), (half *)b11.data_ptr<at::Half>(),
        (half *)w12.data_ptr<at::Half>(), (half *)b12.data_ptr<at::Half>(),
        (half *)t1.data_ptr<at::Half>(),
        (half *)t2.data_ptr<at::Half>(),
        (half *)x_out.data_ptr<at::Half>(),
        (half *)r_out.data_ptr<at::Half>()
    );
}

std::pair<at::Tensor, at::Tensor> nerf(
    at::Tensor x, at::Tensor d,
    at::Tensor w1, at::Tensor b1,
    at::Tensor w2, at::Tensor b2,
    at::Tensor w3, at::Tensor b3,
    at::Tensor w4, at::Tensor b4,
    at::Tensor w5, at::Tensor b5,
    at::Tensor w6, at::Tensor b6,
    at::Tensor w7, at::Tensor b7,
    at::Tensor w8, at::Tensor b8,
    at::Tensor w9, at::Tensor b9,
    at::Tensor w10, at::Tensor b10,
    at::Tensor w11, at::Tensor b11,
    at::Tensor w12, at::Tensor b12
) {

    at::Tensor t1 = at::zeros({x.size(0), 320}, x.options());
    at::Tensor t2 = at::zeros({x.size(0), 320}, x.options());
    at::Tensor x_out = at::zeros({x.size(0), 16}, x.options());
    at::Tensor r_out = at::zeros({x.size(0), 16}, x.options());
    nerf_out(
        x, d,
        w1, b1,
        w2, b2,
        w3, b3,
        w4, b4,
        w5, b5,
        w6, b6,
        w7, b7,
        w8, b8,
        w9, b9,
        w10, b10,
        w11, b11,
        w12, b12,
        t1,
        t2,
        x_out,
        r_out
    );

    return std::make_pair(x_out, r_out);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("nerf", &nerf, "nerf");
    m.def("nerf_out", &nerf_out, "nerf_out");
}

