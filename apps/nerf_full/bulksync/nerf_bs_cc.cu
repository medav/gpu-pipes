

#include <hip/hip_fp16.h>

#include "wrapper_utils.cuh"
#include "nerf_bs.cuh"
#include "bulksync_gemm.cuh"
#include "utils.cuh"

int main(int argc, char * argv[]) {
    const size_t NI = argc > 1 ? std::atoi(argv[1]) : 1000;
    printf("NI: %zu\n", NI);

    ALLOC_TENSOR_2D(x, MM, 64)
    ALLOC_TENSOR_2D(d, MM, 32)

    ALLOC_LINEAR_WEIGHTS(l1, 64, 256)
    ALLOC_LINEAR_WEIGHTS(l2, 256, 256)
    ALLOC_LINEAR_WEIGHTS(l3, 256, 256)
    ALLOC_LINEAR_WEIGHTS(l4, 256, 256)
    ALLOC_LINEAR_WEIGHTS(l5, 256, 256)
    ALLOC_LINEAR_WEIGHTS(l6, 256, 256)
    ALLOC_LINEAR_WEIGHTS(l7, 256, 256)
    ALLOC_LINEAR_WEIGHTS(l8, 256, 256)
    ALLOC_LINEAR_WEIGHTS(l9, 256, 16)
    ALLOC_LINEAR_WEIGHTS(l10, 256, 256)
    ALLOC_LINEAR_WEIGHTS(l11, 288, 128)
    ALLOC_LINEAR_WEIGHTS(l12, 128, 16)

    ALLOC_TENSOR_2D(t1, MM, 320)
    ALLOC_TENSOR_2D(t2, MM, 320)
    ALLOC_TENSOR_2D(rgb_out, MM, 16)
    ALLOC_TENSOR_2D(r_out, MM, 16)


    float time_ms = cuda_time_kernel_ms([&]() {
        for (size_t i = 0; i < NI; i++) {
            nerf_bs<MM>(
                x, d,
                l1_w, l1_b,
                l2_w, l2_b,
                l3_w, l3_b,
                l4_w, l4_b,
                l5_w, l5_b,
                l6_w, l6_b,
                l7_w, l7_b,
                l8_w, l8_b,
                l9_w, l9_b,
                l10_w, l10_b,
                l11_w, l11_b,
                l12_w, l12_b,
                t1, t2,
                rgb_out, r_out
            );
        }
    });

    printf("Avg latency: %f ms\n", time_ms / (float)NI);

    const float flops = MM * (
        256 * 60 +
        256 * 256 +
        256 * 256 +
        256 * 256 +
        256 * 256 +
        256 * 316 +
        256 * 256 +
        256 * 256 +
        256 * 256 +
        128 * 280 +
        3 * 128 +
        1 * 256
    ) * 2.0f;

    printf("GFLOPS: %f\n", NI * flops / (time_ms * 1e-3f) / 1e9f);
}
