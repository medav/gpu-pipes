

#include <torch/extension.h>
#include <ATen/ATen.h>

#include <hip/hip_fp16.h>

#include "pipes.cuh"
#include "pipe_gemm.cuh"
#include "pipe_gemm_bias.cuh"
#include "pipe_gemm_bias_relu.cuh"
#include "pipe_layer_norm.cuh"

#include "utils.cuh"

#include "nerf_full_pl.cuh"

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)



void nerf_full_out(
    at::Tensor x,
    at::Tensor d,
    at::Tensor w1, at::Tensor b1,
    at::Tensor w2, at::Tensor b2,
    at::Tensor w3, at::Tensor b3,
    at::Tensor w4, at::Tensor b4,
    at::Tensor w5, at::Tensor b5,
    at::Tensor w6, at::Tensor b6,
    at::Tensor w7, at::Tensor b7,
    at::Tensor w8, at::Tensor b8,
    at::Tensor w9, at::Tensor b9,
    at::Tensor w10, at::Tensor b10,
    at::Tensor w11, at::Tensor b11,
    at::Tensor w12, at::Tensor b12,
    at::Tensor out_r,
    at::Tensor out_rgb
) {
    CHECK_INPUT(x);
    CHECK_INPUT(w1); CHECK_INPUT(b1);
    CHECK_INPUT(w2); CHECK_INPUT(b2);
    CHECK_INPUT(w3); CHECK_INPUT(b3);
    CHECK_INPUT(w4); CHECK_INPUT(b4);
    CHECK_INPUT(w5); CHECK_INPUT(b5);
    CHECK_INPUT(w6); CHECK_INPUT(b6);
    CHECK_INPUT(w7); CHECK_INPUT(b7);
    CHECK_INPUT(w8); CHECK_INPUT(b8);
    CHECK_INPUT(w9); CHECK_INPUT(b9);
    CHECK_INPUT(w10); CHECK_INPUT(b10);
    CHECK_INPUT(w11); CHECK_INPUT(b11);
    CHECK_INPUT(w12); CHECK_INPUT(b12);

    assert(x.size(1) == 64);
    assert(w1.size(0) == 64 && w1.size(1) == 256 && b1.size(0) == 256);
    assert(w2.size(0) == 256 && w2.size(1) == 256 && b2.size(0) == 256);
    assert(w3.size(0) == 256 && w3.size(1) == 256 && b3.size(0) == 256);
    assert(w4.size(0) == 256 && w4.size(1) == 256 && b4.size(0) == 256);
    assert(w5.size(0) == 256 && w5.size(1) == 256 && b5.size(0) == 256);
    assert(w6.size(0) == 320 && w6.size(1) == 256 && b6.size(0) == 256);
    assert(w7.size(0) == 256 && w7.size(1) == 256 && b7.size(0) == 256);
    assert(w8.size(0) == 256 && w8.size(1) == 256 && b8.size(0) == 256);
    assert(w9.size(0) == 256 && w9.size(1) == 64 && b9.size(0) == 64);
    assert(w10.size(0) == 256 && w10.size(1) == 256 && b10.size(0) == 256);
    assert(w11.size(0) == 288 && w11.size(1) == 128 && b11.size(0) == 128);
    assert(w12.size(0) == 128 && w12.size(1) == 64 && b12.size(0) == 64);
    assert(x.size(0) % NerfFullMlp::n_rows == 0);

    dim3 grid(NerfFullMlp::n_cols, NerfFullMlp::n_rows);
    dim3 block(32, num_warps);

    configure_smem_once();

    nerf_full_device<<<grid, block, max_smem>>>(
        x.size(0),
        (half *)x.data_ptr<at::Half>(),
        (half *)d.data_ptr<at::Half>(),
        (half *)w1.data_ptr<at::Half>(), (half *)b1.data_ptr<at::Half>(),
        (half *)w2.data_ptr<at::Half>(), (half *)b2.data_ptr<at::Half>(),
        (half *)w3.data_ptr<at::Half>(), (half *)b3.data_ptr<at::Half>(),
        (half *)w4.data_ptr<at::Half>(), (half *)b4.data_ptr<at::Half>(),
        (half *)w5.data_ptr<at::Half>(), (half *)b5.data_ptr<at::Half>(),
        (half *)w6.data_ptr<at::Half>(), (half *)b6.data_ptr<at::Half>(),
        (half *)w7.data_ptr<at::Half>(), (half *)b7.data_ptr<at::Half>(),
        (half *)w8.data_ptr<at::Half>(), (half *)b8.data_ptr<at::Half>(),
        (half *)w9.data_ptr<at::Half>(), (half *)b9.data_ptr<at::Half>(),
        (half *)w10.data_ptr<at::Half>(), (half *)b10.data_ptr<at::Half>(),
        (half *)w11.data_ptr<at::Half>(), (half *)b11.data_ptr<at::Half>(),
        (half *)w12.data_ptr<at::Half>(), (half *)b12.data_ptr<at::Half>(),
        (half *)out_r.data_ptr<at::Half>(),
        (half *)out_rgb.data_ptr<at::Half>(),
        global_queue_space()
    );
}


std::pair<at::Tensor, at::Tensor> nerf_full(
    at::Tensor x,
    at::Tensor d,
    at::Tensor w1, at::Tensor b1,
    at::Tensor w2, at::Tensor b2,
    at::Tensor w3, at::Tensor b3,
    at::Tensor w4, at::Tensor b4,
    at::Tensor w5, at::Tensor b5,
    at::Tensor w6, at::Tensor b6,
    at::Tensor w7, at::Tensor b7,
    at::Tensor w8, at::Tensor b8,
    at::Tensor w9, at::Tensor b9,
    at::Tensor w10, at::Tensor b10,
    at::Tensor w11, at::Tensor b11,
    at::Tensor w12, at::Tensor b12
) {
    at::Tensor out_r = at::zeros({x.size(0), 64}, x.options());
    at::Tensor out_rgb = at::zeros({x.size(0), 64}, x.options());
    nerf_full_out(
        x, d,
        w1, b1,
        w2, b2,
        w3, b3,
        w4, b4,
        w5, b5,
        w6, b6,
        w7, b7,
        w8, b8,
        w9, b9,
        w10, b10,
        w11, b11,
        w12, b12,
        out_r, out_rgb
    );
    return {out_r, out_rgb};
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("nerf_full", &nerf_full, "nerf_full");
    m.def("nerf_full_out", &nerf_full_out, "nerf_full_out");
}

