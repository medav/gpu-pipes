

#include <torch/extension.h>
#include <ATen/ATen.h>

#include <hip/hip_fp16.h>

#include "test_mlp_pl.cuh"
#include "utils.cuh"

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)


inline typename TestMlp::Queues * global_queue_space() {
    static typename TestMlp::Queues * qs_dev = nullptr;

    if (qs_dev != nullptr) return qs_dev;

    cudaErrCheck(hipMalloc(&qs_dev, TestMlp::n_rows * sizeof(*qs_dev)));
    cudaErrCheck(hipMemset(qs_dev, 0, TestMlp::n_rows * sizeof(*qs_dev)));

    pin_memory(qs_dev, TestMlp::n_rows * sizeof(*qs_dev));

    return qs_dev;
}

inline void configure_smem_once() {
    static bool configured = false;
    if (configured) return;
    configure_smem((const void *)testmlp_device, max_smem);
    configured = true;
}


void testmlp_out(
    at::Tensor x,     // [M, 128]
    at::Tensor w1,    // [128, 128]
    at::Tensor b1,    // [128]
    at::Tensor w2,    // [128, 128]
    at::Tensor b2,    // [128]
    at::Tensor w3,    // [128, 128]
    at::Tensor b3,    // [128]
    at::Tensor out    // [M, 128]
) {
    CHECK_INPUT(x);
    CHECK_INPUT(w1);
    CHECK_INPUT(b1);
    CHECK_INPUT(w2);
    CHECK_INPUT(b2);
    CHECK_INPUT(w3);
    CHECK_INPUT(b3);
    CHECK_INPUT(out);

    assert(x.size(1) == 128);
    assert(w1.size(0) == 128 && w1.size(1) == 128 && b1.size(0) == 128);
    assert(w2.size(0) == 128 && w2.size(1) == 128 && b2.size(0) == 128);
    assert(w3.size(0) == 128 && w3.size(1) == 128 && b3.size(0) == 128);

    dim3 grid(TestMlp::n_cols, TestMlp::n_rows);
    dim3 block(32, num_warps);

    configure_smem_once();

    cuda_check_kernel_call([&]() {
        testmlp_device<<<grid, block, max_smem>>>(
            x.size(0),
            (half *)x.data_ptr<at::Half>(),
            (half *)w1.data_ptr<at::Half>(),
            (half *)b1.data_ptr<at::Half>(),
            (half *)w2.data_ptr<at::Half>(),
            (half *)b2.data_ptr<at::Half>(),
            (half *)w3.data_ptr<at::Half>(),
            (half *)b3.data_ptr<at::Half>(),
            (half *)out.data_ptr<at::Half>(),
            global_queue_space()
        );
    });
}

at::Tensor testmlp(
    at::Tensor x,     // [M, 384]
    at::Tensor w1,    // [384, 128]
    at::Tensor b1,    // [128]
    at::Tensor w2,    // [128, 128]
    at::Tensor b2,    // [128]
    at::Tensor w3,    // [128, 128]
    at::Tensor b3     // [128]
) {
    at::Tensor out = at::zeros({x.size(0), 128}, x.options());
    testmlp_out(x, w1, b1, w2, b2, w3, b3, out);
    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("testmlp", &testmlp, "testmlp");
    m.def("testmlp_out", &testmlp_out, "testmlp_out");
}

