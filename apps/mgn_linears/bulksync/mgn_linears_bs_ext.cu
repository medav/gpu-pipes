#include "hip/hip_runtime.h"


#include <torch/extension.h>
#include <ATen/ATen.h>

#include <hip/hip_fp16.h>

#include "mgn_linears_bs.cuh"
#include "bulksync_gemm.cuh"
#include "utils.cuh"

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)


void mgn_linears_out(
    at::Tensor x,     // [M, 384]
    at::Tensor w1,    // [128, 128]
    at::Tensor b1,    // [128]
    at::Tensor w2,    // [128, 128]
    at::Tensor b2,    // [128]
    at::Tensor w3,    // [128, 128]
    at::Tensor b3,    // [128]
    at::Tensor y1,    // [M, 128]
    at::Tensor y2,    // [M, 128]
    at::Tensor out    // [M, 128]
) {
    CHECK_INPUT(x);
    CHECK_INPUT(w1);
    CHECK_INPUT(b1);
    CHECK_INPUT(w2);
    CHECK_INPUT(b2);
    CHECK_INPUT(w3);
    CHECK_INPUT(b3);
    CHECK_INPUT(y1);
    CHECK_INPUT(y2);
    CHECK_INPUT(out);

    assert(x.size(0) == MM);
    assert(x.size(1) == 384);
    assert(w1.size(0) == 384 && w1.size(1) == 128 && b1.size(0) == 128);
    assert(w2.size(0) == 128 && w2.size(1) == 128 && b2.size(0) == 128);
    assert(w3.size(0) == 128 && w3.size(1) == 128 && b3.size(0) == 128);

    mgn_linears_bs<MM, DD>(
        (half *)x.data_ptr<at::Half>(),
        (half *)w1.data_ptr<at::Half>(),
        (half *)b1.data_ptr<at::Half>(),
        (half *)w2.data_ptr<at::Half>(),
        (half *)b2.data_ptr<at::Half>(),
        (half *)w3.data_ptr<at::Half>(),
        (half *)b3.data_ptr<at::Half>(),
        (half *)y1.data_ptr<at::Half>(),
        (half *)y2.data_ptr<at::Half>(),
        (half *)out.data_ptr<at::Half>()
    );
}

at::Tensor mgn_linears(
    at::Tensor x,     // [M, 384]
    at::Tensor w1,    // [384, 128]
    at::Tensor b1,    // [128]
    at::Tensor w2,    // [128, 128]
    at::Tensor b2,    // [128]
    at::Tensor w3,    // [128, 128]
    at::Tensor b3     // [128]
) {

    at::Tensor y1 = at::zeros({x.size(0), 128}, x.options());
    at::Tensor y2 = at::zeros({x.size(0), 128}, x.options());
    at::Tensor out = at::zeros({x.size(0), 128}, x.options());
    mgn_linears_out(x, w1, b1, w2, b2, w3, b3, y1, y2, out);
    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("mgn_linears", &mgn_linears, "mgn_linears");
    m.def("mgn_linears_out", &mgn_linears_out, "mgn_linears_out");
}

