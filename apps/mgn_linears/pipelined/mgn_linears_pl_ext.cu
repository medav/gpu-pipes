

#include <torch/extension.h>
#include <ATen/ATen.h>

#include <hip/hip_fp16.h>

#include "pipes.cuh"
#include "pipe_gemm.cuh"
#include "pipe_gemm_bias.cuh"
#include "pipe_gemm_bias_relu.cuh"
#include "pipe_layer_norm.cuh"

#include "utils.cuh"

#include "mgn_linears_pl.cuh"

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)



void mgn_linears_out(
    at::Tensor x,     // [M, 384]
    at::Tensor w1,    // [384, 128]
    at::Tensor b1,    // [128]
    at::Tensor w2,    // [128, 128]
    at::Tensor b2,    // [128]
    at::Tensor w3,    // [128, 128]
    at::Tensor b3,    // [128]
    at::Tensor out    // [M, 128]
) {
    CHECK_INPUT(x);
    CHECK_INPUT(w1);
    CHECK_INPUT(b1);
    CHECK_INPUT(w2);
    CHECK_INPUT(b2);
    CHECK_INPUT(w3);
    CHECK_INPUT(b3);

    assert(x.size(1) == 384);
    assert(w1.size(0) == 384 && w1.size(1) == 128 && b1.size(0) == 128);
    assert(w2.size(0) == 128 && w2.size(1) == 128 && b2.size(0) == 128);
    assert(w3.size(0) == 128 && w3.size(1) == 128 && b3.size(0) == 128);

    dim3 grid(MgnLinears::n_cols, MgnLinears::n_rows);
    dim3 block(32, num_warps);

    configure_smem_once();

    linears_device<<<grid, block, max_smem>>>(
        x.size(0),
        (half *)x.data_ptr<at::Half>(),
        (half *)w1.data_ptr<at::Half>(),
        (half *)b1.data_ptr<at::Half>(),
        (half *)w2.data_ptr<at::Half>(),
        (half *)b2.data_ptr<at::Half>(),
        (half *)w3.data_ptr<at::Half>(),
        (half *)b3.data_ptr<at::Half>(),
        (half *)out.data_ptr<at::Half>(),
        global_queue_space()
    );
}


at::Tensor mgn_linears(
    at::Tensor x,     // [M, 384]
    at::Tensor w1,    // [384, 128]
    at::Tensor b1,    // [128]
    at::Tensor w2,    // [128, 128]
    at::Tensor b2,    // [128]
    at::Tensor w3,    // [128, 128]
    at::Tensor b3    // [128]
) {
    at::Tensor out = at::zeros({x.size(0), 128}, x.options());
    mgn_linears_out(x, w1, b1, w2, b2, w3, b3, out);
    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("mgn_linears", &mgn_linears, "mgn_linears");
    m.def("mgn_linears_out", &mgn_linears_out, "mgn_linears_out");
}

