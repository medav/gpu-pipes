

#include <torch/extension.h>
#include <ATen/ATen.h>

#include <hip/hip_fp16.h>

#include "pipes.cuh"
#include "pipe_gemm.cuh"
#include "pipe_gemm_bias.cuh"
#include "pipe_gemm_bias_relu.cuh"
#include "pipe_layer_norm.cuh"

#include "utils.cuh"

#include "dlrm_topmlp_pl.cuh"

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)



void dlrm_topmlp_out(
    at::Tensor x,
    at::Tensor w1, at::Tensor b1,
    at::Tensor w2, at::Tensor b2,
    at::Tensor w3, at::Tensor b3,
    at::Tensor w4, at::Tensor b4,
    at::Tensor w5, at::Tensor b5,
    at::Tensor out
) {
    CHECK_INPUT(x);
    CHECK_INPUT(w1);
    CHECK_INPUT(b1);
    CHECK_INPUT(w2);
    CHECK_INPUT(b2);
    CHECK_INPUT(w3);
    CHECK_INPUT(b3);

    assert(x.size(1) == DlrmTopMlp::d0);
    assert(w1.size(0) == DlrmTopMlp::d0 && w1.size(1) == DlrmTopMlp::d1 && b1.size(0) == DlrmTopMlp::d1);
    assert(w2.size(0) == DlrmTopMlp::d1 && w2.size(1) == DlrmTopMlp::d2 && b2.size(0) == DlrmTopMlp::d2);
    assert(w3.size(0) == DlrmTopMlp::d2 && w3.size(1) == DlrmTopMlp::d3 && b3.size(0) == DlrmTopMlp::d3);
    assert(w4.size(0) == DlrmTopMlp::d3 && w4.size(1) == DlrmTopMlp::d4 && b4.size(0) == DlrmTopMlp::d4);
    assert(w5.size(0) == DlrmTopMlp::d4 && w5.size(1) == DlrmTopMlp::d5 && b5.size(0) == DlrmTopMlp::d5);
    assert(x.size(0) % DlrmTopMlp::n_rows == 0);

    dim3 grid(DlrmTopMlp::n_cols, DlrmTopMlp::n_rows);
    dim3 block(32, num_warps);

    configure_smem_once();

    dlrm_topmlp_device<<<grid, block, max_smem>>>(
        x.size(0),
        (half *)x.data_ptr<at::Half>(),
        (half *)w1.data_ptr<at::Half>(),
        (half *)b1.data_ptr<at::Half>(),
        (half *)w2.data_ptr<at::Half>(),
        (half *)b2.data_ptr<at::Half>(),
        (half *)w3.data_ptr<at::Half>(),
        (half *)b3.data_ptr<at::Half>(),
        (half *)w4.data_ptr<at::Half>(),
        (half *)b4.data_ptr<at::Half>(),
        (half *)w5.data_ptr<at::Half>(),
        (half *)b5.data_ptr<at::Half>(),
        (half *)out.data_ptr<at::Half>(),
        global_queue_space()
    );
}


at::Tensor dlrm_topmlp(
    at::Tensor x,
    at::Tensor w1, at::Tensor b1,
    at::Tensor w2, at::Tensor b2,
    at::Tensor w3, at::Tensor b3,
    at::Tensor w4, at::Tensor b4,
    at::Tensor w5, at::Tensor b5
) {
    at::Tensor out = at::zeros({x.size(0), DlrmTopMlp::d5}, x.options());
    dlrm_topmlp_out(x, w1, b1, w2, b2, w3, b3, w4, b4, w5, b5, out);
    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("dlrm_topmlp", &dlrm_topmlp, "dlrm_topmlp");
    m.def("dlrm_topmlp_out", &dlrm_topmlp_out, "dlrm_topmlp_out");
}

