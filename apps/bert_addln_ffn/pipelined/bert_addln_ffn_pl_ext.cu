

#include <torch/extension.h>
#include <ATen/ATen.h>

#include <hip/hip_fp16.h>

#include "bert_addln_ffn_pl.cuh"
#include "bulksync_gemm.cuh"
#include "utils.cuh"

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)


void bert_addln_ffn_out(
    at::Tensor attn_out,
    at::Tensor x,
    at::Tensor w0, at::Tensor b0,
    at::Tensor w1, at::Tensor b1,
    at::Tensor w2, at::Tensor b2,
    at::Tensor ga0, at::Tensor be0,
    at::Tensor ga2, at::Tensor be2,
    at::Tensor out
) {
    CHECK_INPUT(x);
    CHECK_INPUT(w0); CHECK_INPUT(b0);
    CHECK_INPUT(w1); CHECK_INPUT(b1);
    CHECK_INPUT(w2); CHECK_INPUT(b2);
    CHECK_INPUT(ga0); CHECK_INPUT(be0);
    CHECK_INPUT(ga2); CHECK_INPUT(be2);

    assert(attn_out.size(1) == 128);
    assert(x.size(1) == 128);
    assert(w0.size(0) == 128 && w0.size(1) == 128 && b0.size(0) == 128);
    assert(w1.size(0) == 128 && w1.size(1) == 512 && b1.size(0) == 512);
    assert(w2.size(0) == 512 && w2.size(1) == 128 && b2.size(0) == 128);
    assert(ga0.size(0) == 128 && be0.size(0) == 128);
    assert(ga2.size(0) == 128 && be2.size(0) == 128);

    dim3 grid(BertFfn::n_cols, BertFfn::n_rows);
    dim3 block(32, num_warps);

    configure_smem_once();

    bert_ffn_device<<<grid, block, max_smem>>>(
        x.size(0),
        (half *)attn_out.data_ptr<at::Half>(), (half *)x.data_ptr<at::Half>(),
        (half *)w0.data_ptr<at::Half>(), (half *)b0.data_ptr<at::Half>(),
        (half *)w1.data_ptr<at::Half>(), (half *)b1.data_ptr<at::Half>(),
        (half *)w2.data_ptr<at::Half>(), (half *)b2.data_ptr<at::Half>(),
        (half *)ga0.data_ptr<at::Half>(), (half *)be0.data_ptr<at::Half>(),
        (half *)ga2.data_ptr<at::Half>(), (half *)be2.data_ptr<at::Half>(),
        (half *)out.data_ptr<at::Half>(),
        global_queue_space()
    );
}

at::Tensor bert_addln_ffn(
    at::Tensor attn_out,
    at::Tensor x,
    at::Tensor w0, at::Tensor b0,
    at::Tensor w1, at::Tensor b1,
    at::Tensor w2, at::Tensor b2,
    at::Tensor ga0, at::Tensor be0,
    at::Tensor ga2, at::Tensor be2
) {

    at::Tensor out = at::zeros({x.size(0), 128}, x.options());
    bert_addln_ffn_out(attn_out, x, w0, b0, w1, b1, w2, b2, ga0, be0, ga2, be0, out);
    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("bert_addln_ffn", &bert_addln_ffn, "bert_addln_ffn");
    m.def("bert_addln_ffn_out", &bert_addln_ffn_out, "bert_addln_ffn_out");
}

