

#include <hip/hip_fp16.h>

#include "wrapper_utils.cuh"
#include "dlrm_botmlp_bs.cuh"
#include "bulksync_gemm.cuh"
#include "utils.cuh"

int main(int argc, char * argv[]) {
    const size_t NI = argc > 1 ? std::atoi(argv[1]) : 1000;
    printf("NI: %zu\n", NI);

    ALLOC_TENSOR_2D(x, MM, 128)
    ALLOC_LINEAR_WEIGHTS(l1, 32, 512)
    ALLOC_LINEAR_WEIGHTS(l2, 512, 256)
    ALLOC_LINEAR_WEIGHTS(l3, 256, 128)
    ALLOC_TENSOR_2D(t1, MM, 128)
    ALLOC_TENSOR_2D(t2, MM, 128)
    ALLOC_TENSOR_2D(out, MM, 128)

    float time_ms = cuda_time_kernel_ms([&]() {
        for (size_t i = 0; i < NI; i++) {
            dlrm_botmlp_bs<MM>(
                x,
                l1_w, l1_b,
                l2_w, l2_b,
                l3_w, l3_b,
                t1, t2, out);
        }
    });

    printf("Avg latency: %f ms\n", time_ms / (float)NI);

    const float flops = MM * (512 * 13 + 256 * 512 + 128 * 256) * 2.0f;

    printf("GFLOPS: %f\n", NI * flops / (time_ms * 1e-3f) / 1e9f);
}
