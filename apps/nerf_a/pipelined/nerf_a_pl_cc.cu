

#include <hip/hip_fp16.h>

#include "wrapper_utils.cuh"
#include "nerf_a_pl.cuh"
#include "bulksync_gemm.cuh"
#include "utils.cuh"

const size_t MM = 65536;

int main(int argc, char * argv[]) {
    const size_t NI = argc > 1 ? std::atoi(argv[1]) : 1000;
    printf("NI: %zu\n", NI);


    ALLOC_TENSOR_2D(x, MM, 64)

    ALLOC_LINEAR_WEIGHTS(l1, 64, 256)
    ALLOC_LINEAR_WEIGHTS(l2, 256, 256)
    ALLOC_LINEAR_WEIGHTS(l3, 256, 256)
    ALLOC_LINEAR_WEIGHTS(l4, 256, 256)
    ALLOC_LINEAR_WEIGHTS(l5, 256, 256)
    ALLOC_LINEAR_WEIGHTS(l6, 256, 256)
    ALLOC_LINEAR_WEIGHTS(l7, 256, 256)
    ALLOC_LINEAR_WEIGHTS(l8, 256, 256)

    ALLOC_TENSOR_2D(out, MM, 256)

    dim3 grid(NerfAMlp::n_cols, NerfAMlp::n_rows);
    dim3 block(32, num_warps);

    configure_smem_once();

    float time_ms = cuda_time_kernel_ms([&]() {
        for (size_t i = 0; i < NI; i++) {
            nerf_a_device<<<grid, block, max_smem>>>(
                MM,
                x,
                l1_w, l1_b,
                l2_w, l2_b,
                l3_w, l3_b,
                l4_w, l4_b,
                l5_w, l5_b,
                l6_w, l6_b,
                l7_w, l7_b,
                l8_w, l8_b,
                out,
                global_queue_space()
            );
        }
    });

    printf("Avg latency: %f ms\n", time_ms / (float)NI);

    const float flops = MM * (
        256 * 60 +
        256 * 256 +
        256 * 256 +
        256 * 256 +
        256 * 256 +
        256 * 316 +
        256 * 256 +
        256 * 256
    ) * 2.0f;

    printf("FLOPS: %f\n", flops);

    printf("GFLOPS: %f\n", (float)NI * flops / (time_ms * 1e-3f) / 1e9f);
}
