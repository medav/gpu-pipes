

#include <torch/extension.h>
#include <ATen/ATen.h>

#include <hip/hip_fp16.h>

#include "nerf_a_bs.cuh"
#include "bulksync_gemm.cuh"
#include "utils.cuh"

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)


void nerf_a_out(
    at::Tensor x,
    at::Tensor w1, at::Tensor b1,
    at::Tensor w2, at::Tensor b2,
    at::Tensor w3, at::Tensor b3,
    at::Tensor w4, at::Tensor b4,
    at::Tensor w5, at::Tensor b5,
    at::Tensor w6, at::Tensor b6,
    at::Tensor w7, at::Tensor b7,
    at::Tensor w8, at::Tensor b8,
    at::Tensor t1,
    at::Tensor t2,
    at::Tensor out
) {
    CHECK_INPUT(x);
    CHECK_INPUT(w1); CHECK_INPUT(b1);
    CHECK_INPUT(w2); CHECK_INPUT(b2);
    CHECK_INPUT(w3); CHECK_INPUT(b3);
    CHECK_INPUT(w4); CHECK_INPUT(b4);
    CHECK_INPUT(w5); CHECK_INPUT(b5);
    CHECK_INPUT(w6); CHECK_INPUT(b6);
    CHECK_INPUT(w7); CHECK_INPUT(b7);
    CHECK_INPUT(w8); CHECK_INPUT(b8);

    assert(x.size(0) == MM);
    assert(x.size(1) == 64);

    assert(w1.size(0) == 64 && w1.size(1) == 256 && b1.size(0) == 256);
    assert(w2.size(0) == 256 && w2.size(1) == 256 && b2.size(0) == 256);
    assert(w3.size(0) == 256 && w3.size(1) == 256 && b3.size(0) == 256);
    assert(w4.size(0) == 256 && w4.size(1) == 256 && b4.size(0) == 256);
    assert(w5.size(0) == 256 && w5.size(1) == 256 && b5.size(0) == 256);
    assert(w6.size(0) == 320 && w6.size(1) == 256 && b6.size(0) == 256);
    assert(w7.size(0) == 256 && w7.size(1) == 256 && b7.size(0) == 256);
    assert(w8.size(0) == 256 && w8.size(1) == 256 && b8.size(0) == 256);

    nerf_a_bs<MM>(
        (half *)x.data_ptr<at::Half>(),
        (half *)w1.data_ptr<at::Half>(), (half *)b1.data_ptr<at::Half>(),
        (half *)w2.data_ptr<at::Half>(), (half *)b2.data_ptr<at::Half>(),
        (half *)w3.data_ptr<at::Half>(), (half *)b3.data_ptr<at::Half>(),
        (half *)w4.data_ptr<at::Half>(), (half *)b4.data_ptr<at::Half>(),
        (half *)w5.data_ptr<at::Half>(), (half *)b5.data_ptr<at::Half>(),
        (half *)w6.data_ptr<at::Half>(), (half *)b6.data_ptr<at::Half>(),
        (half *)w7.data_ptr<at::Half>(), (half *)b7.data_ptr<at::Half>(),
        (half *)w8.data_ptr<at::Half>(), (half *)b8.data_ptr<at::Half>(),
        (half *)t1.data_ptr<at::Half>(),
        (half *)t2.data_ptr<at::Half>(),
        (half *)out.data_ptr<at::Half>()
    );
}

at::Tensor nerf_a(
    at::Tensor x,
    at::Tensor w1, at::Tensor b1,
    at::Tensor w2, at::Tensor b2,
    at::Tensor w3, at::Tensor b3,
    at::Tensor w4, at::Tensor b4,
    at::Tensor w5, at::Tensor b5,
    at::Tensor w6, at::Tensor b6,
    at::Tensor w7, at::Tensor b7,
    at::Tensor w8, at::Tensor b8
) {

    at::Tensor t1 = at::zeros({x.size(0), 320}, x.options());
    at::Tensor t2 = at::zeros({x.size(0), 320}, x.options());
    at::Tensor out = at::zeros({x.size(0), 256}, x.options());
    nerf_a_out(
        x,
        w1, b1,
        w2, b2,
        w3, b3,
        w4, b4,
        w5, b5,
        w6, b6,
        w7, b7,
        w8, b8,
        t1,
        t2,
        out
    );

    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("nerf_a", &nerf_a, "nerf_a");
    m.def("nerf_a_out", &nerf_a_out, "nerf_a_out");
}

