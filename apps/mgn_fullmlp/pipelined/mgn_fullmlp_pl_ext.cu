

#include <torch/extension.h>
#include <ATen/ATen.h>

#include <hip/hip_fp16.h>

#include "pipes.cuh"
#include "pipe_gemm.cuh"
#include "pipe_gemm_bias.cuh"
#include "pipe_gemm_bias_relu.cuh"
#include "pipe_layer_norm.cuh"

#include "utils.cuh"

#include "mgn_fullmlp_pl.cuh"

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)



void mgn_fullmlp_out(
    at::Tensor x,     // [M, 384]
    at::Tensor w1,    // [384, 128]
    at::Tensor b1,    // [128]
    at::Tensor w2,    // [128, 128]
    at::Tensor b2,    // [128]
    at::Tensor w3,    // [128, 128]
    at::Tensor b3,    // [128]
    at::Tensor gamma, // [128]
    at::Tensor beta,  // [128]
    at::Tensor out    // [M, 128]
) {
    CHECK_INPUT(x);
    CHECK_INPUT(w1);
    CHECK_INPUT(b1);
    CHECK_INPUT(w2);
    CHECK_INPUT(b2);
    CHECK_INPUT(w3);
    CHECK_INPUT(b3);

    assert(x.size(1) == 384);
    assert(w1.size(0) == 384 && w1.size(1) == 128 && b1.size(0) == 128);
    assert(w2.size(0) == 128 && w2.size(1) == 128 && b2.size(0) == 128);
    assert(w3.size(0) == 128 && w3.size(1) == 128 && b3.size(0) == 128);
    assert(gamma.size(0) == 128 && beta.size(0) == 128);

    dim3 grid(MgnFullMlp::n_cols, MgnFullMlp::n_rows);
    dim3 block(32, num_warps);

    configure_smem_once();

    fullmlp_device<<<grid, block, max_smem>>>(
        x.size(0),
        (half *)x.data_ptr<at::Half>(),
        (half *)w1.data_ptr<at::Half>(),
        (half *)b1.data_ptr<at::Half>(),
        (half *)w2.data_ptr<at::Half>(),
        (half *)b2.data_ptr<at::Half>(),
        (half *)w3.data_ptr<at::Half>(),
        (half *)b3.data_ptr<at::Half>(),
        (half *)gamma.data_ptr<at::Half>(),
        (half *)beta.data_ptr<at::Half>(),
        (half *)out.data_ptr<at::Half>(),
        global_queue_space()
    );
}


at::Tensor mgn_fullmlp(
    at::Tensor x,     // [M, 384]
    at::Tensor w1,    // [384, 128]
    at::Tensor b1,    // [128]
    at::Tensor w2,    // [128, 128]
    at::Tensor b2,    // [128]
    at::Tensor w3,    // [128, 128]
    at::Tensor b3,    // [128]
    at::Tensor gamma, // [128]
    at::Tensor beta   // [128]
) {
    at::Tensor out = at::zeros({x.size(0), 128}, x.options());
    mgn_fullmlp_out(x, w1, b1, w2, b2, w3, b3, gamma, beta, out);
    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("mgn_fullmlp", &mgn_fullmlp, "mgn_fullmlp");
    m.def("mgn_fullmlp_out", &mgn_fullmlp_out, "mgn_fullmlp_out");
}

