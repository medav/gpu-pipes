

#include <hip/hip_fp16.h>

#include "test_mlp_bs.cuh"
#include "bulksync_gemm.cuh"
#include "utils.cuh"

int main(int argc, char * argv[]) {
    const size_t NI = argc > 1 ? std::atoi(argv[1]) : 1000;
    printf("NI: %zu\n", NI);

    half * x_dev = nullptr;
    half * w1_dev = nullptr;
    half * b1_dev = nullptr;
    half * w2_dev = nullptr;
    half * b2_dev = nullptr;
    half * w3_dev = nullptr;
    half * b3_dev = nullptr;
    half * gamma_dev = nullptr;
    half * beta_dev = nullptr;
    half * y1_dev = nullptr;
    half * y2_dev = nullptr;
    half * y3_dev = nullptr;
    half * out_dev = nullptr;

    cudaErrCheck(hipMalloc(&x_dev, MM * DD * 3 * sizeof(*x_dev)));
    cudaErrCheck(hipMalloc(&w1_dev, DD * DD * 3 * sizeof(*x_dev)));
    cudaErrCheck(hipMalloc(&b1_dev, DD * sizeof(*x_dev)));
    cudaErrCheck(hipMalloc(&w2_dev, DD * DD * sizeof(*x_dev)));
    cudaErrCheck(hipMalloc(&b2_dev, DD * sizeof(*x_dev)));
    cudaErrCheck(hipMalloc(&w3_dev, DD * DD * sizeof(*x_dev)));
    cudaErrCheck(hipMalloc(&b3_dev, DD * sizeof(*x_dev)));
    cudaErrCheck(hipMalloc(&gamma_dev, DD * sizeof(*x_dev)));
    cudaErrCheck(hipMalloc(&beta_dev, DD * sizeof(*x_dev)));
    cudaErrCheck(hipMalloc(&y1_dev, MM * DD * sizeof(*x_dev)));
    cudaErrCheck(hipMalloc(&y2_dev, MM * DD * sizeof(*x_dev)));
    cudaErrCheck(hipMalloc(&y3_dev, MM * DD * sizeof(*x_dev)));
    cudaErrCheck(hipMalloc(&out_dev, MM * DD * sizeof(*x_dev)));


    float time_ms = cuda_time_kernel_ms([&]() {
        for (size_t i = 0; i < NI; i++) {
            mgn_fullmlp_bs<MM, DD>(
                x_dev,
                w1_dev,
                b1_dev,
                w2_dev,
                b2_dev,
                w3_dev,
                b3_dev,
                gamma_dev,
                beta_dev,
                y1_dev,
                y2_dev,
                y3_dev,
                out_dev);
        }
    });

    printf("Avg latency: %f ms\n", time_ms / (float)NI);

    const float flops = MM * (
        DD * DD * 3 +
        DD * DD +
        DD * DD) * 2.0f;

    printf("GFLOPS: %f\n", NI * flops / (time_ms * 1e-3f) / 1e9f);
}
