#include "hip/hip_runtime.h"


#include <torch/extension.h>
#include <ATen/ATen.h>

#include <hip/hip_fp16.h>

#include "mgn_fullmlp_bs.cuh"
#include "bulksync_gemm.cuh"
#include "utils.cuh"

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)


void mgn_fullmlp_out(
    at::Tensor x,     // [M, 384]
    at::Tensor w1,    // [128, 128]
    at::Tensor b1,    // [128]
    at::Tensor w2,    // [128, 128]
    at::Tensor b2,    // [128]
    at::Tensor w3,    // [128, 128]
    at::Tensor b3,    // [128]
    at::Tensor gamma,    // [128]
    at::Tensor beta,    // [128]
    at::Tensor y1,    // [M, 128]
    at::Tensor y2,    // [M, 128]
    at::Tensor y3,    // [M, 128]
    at::Tensor out    // [M, 128]
) {
    CHECK_INPUT(x);
    CHECK_INPUT(w1);
    CHECK_INPUT(b1);
    CHECK_INPUT(w2);
    CHECK_INPUT(b2);
    CHECK_INPUT(w3);
    CHECK_INPUT(b3);
    CHECK_INPUT(gamma);
    CHECK_INPUT(beta);
    CHECK_INPUT(y1);
    CHECK_INPUT(y2);
    CHECK_INPUT(y3);
    CHECK_INPUT(out);

    assert(x.size(0) == MM);
    assert(x.size(1) == 384);
    assert(w1.size(0) == 384 && w1.size(1) == 128 && b1.size(0) == 128);
    assert(w2.size(0) == 128 && w2.size(1) == 128 && b2.size(0) == 128);
    assert(w3.size(0) == 128 && w3.size(1) == 128 && b3.size(0) == 128);
    assert(gamma.size(0) == 128 && beta.size(0) == 128);

    mgn_fullmlp_bs<MM, DD>(
        (half *)x.data_ptr<at::Half>(),
        (half *)w1.data_ptr<at::Half>(),
        (half *)b1.data_ptr<at::Half>(),
        (half *)w2.data_ptr<at::Half>(),
        (half *)b2.data_ptr<at::Half>(),
        (half *)w3.data_ptr<at::Half>(),
        (half *)b3.data_ptr<at::Half>(),
        (half *)gamma.data_ptr<at::Half>(),
        (half *)beta.data_ptr<at::Half>(),
        (half *)y1.data_ptr<at::Half>(),
        (half *)y2.data_ptr<at::Half>(),
        (half *)y3.data_ptr<at::Half>(),
        (half *)out.data_ptr<at::Half>()
    );
}

at::Tensor mgn_fullmlp(
    at::Tensor x,     // [M, 384]
    at::Tensor w1,    // [384, 128]
    at::Tensor b1,    // [128]
    at::Tensor w2,    // [128, 128]
    at::Tensor b2,    // [128]
    at::Tensor w3,    // [128, 128]
    at::Tensor b3,    // [128]
    at::Tensor gamma,    // [128]
    at::Tensor beta    // [128]
) {

    at::Tensor y1 = at::zeros({x.size(0), 128}, x.options());
    at::Tensor y2 = at::zeros({x.size(0), 128}, x.options());
    at::Tensor y3 = at::zeros({x.size(0), 128}, x.options());
    at::Tensor out = at::zeros({x.size(0), 128}, x.options());
    mgn_fullmlp_out(x, w1, b1, w2, b2, w3, b3, gamma, beta, y1, y2, y3, out);
    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("mgn_fullmlp", &mgn_fullmlp, "mgn_fullmlp");
    m.def("mgn_fullmlp_out", &mgn_fullmlp_out, "mgn_fullmlp_out");
}

